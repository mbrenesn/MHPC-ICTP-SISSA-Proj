#include <iostream>

#include <cusp/print.h>

#include "basis.h"
#include "sparse_hamiltonian.h"


int main(int argc, char **argv)
{
  unsigned int l = 18;
  unsigned int n = 9;
  float V = 0.2;
  float t = -1.0;

  Basis basis(l,n);

  // The size of the Hilbert space
  unsigned int bsize = basis.basis_size(); 

  // Create an integer basis in a host memory container
  unsigned long long int *int_basis = new unsigned long long int[bsize];
  basis.construct_int_basis(int_basis);

  //std::cout << "Here's the basis in int notation:" << std::endl;
  //for(unsigned int i=0;i<basis.basis_size();++i) std::cout << int_basis[i] << std::endl;

  // Create an instance of the sparse hamiltonian class to fill the matrix
  // The matrix is using memory space from the host and the construct_hamiltonian_matrix
  // method fills it with the proper values
  SparseHamiltonian sparse_hamiltonian(bsize);

  sparse_hamiltonian.construct_hamiltonian_matrix(V, t, l, n, int_basis);

  // Now the matrix resides in host and device memory. 
  cusp::array2d<VType, DSpace, cusp::row_major> mat(4,4,VType (1.0, 1.0));
  cusp::array2d<VType, DSpace, cusp::row_major> exp_mat(4,4);

  hipblasHandle_t handle;
  if(hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS){
    throw cusp::runtime_exception("hipblasCreate failed");
  }

  sparse_hamiltonian.expm_pade(exp_mat, mat, 4, 6, handle);

  hipblasDestroy(handle);
  delete [] int_basis;
  return 0;
}
